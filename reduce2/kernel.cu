
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

using namespace std;


void check(hipError_t e)
{
	if (e != hipSuccess)
	{
		printf(hipGetErrorString(e));
	}
}



// Kernel function to add the elements of two arrays
__global__
void reduce(int n, float *x, float *y)
{
	int tid = threadIdx.x;
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	for (int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			x[threadId] += x[threadId+s];
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		y[blockIdx.x] = x[threadId];
	}	
}

int main(void)
{
	int N = 1 <<20;
	int reduced_n = N/1024;
	float *x, *y;
	// Allocate Unified Memory � accessible from CPU or GPU
	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, reduced_n * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; ++i) {
		x[i] = i+1;
	}

	// Run kernel on 1M elements on the GPU
	reduce<<<reduced_n, reduced_n >>>(N, x,y);

	 //Run on one block
	reduce<<<1, reduced_n>>>(N,y,y);


	

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	cout << "The final sum is" << y[0];
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}


	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	/*for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	std::cout << "Max error: " << maxError << std::endl;
*/
	// Free memory
	hipFree(x);
	hipFree(y);
	getchar();
	return 0;
}